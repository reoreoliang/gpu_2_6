#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include "stdio.h"

// ��������ÿһ��Ԫ�ؽ���ƽ��

constexpr auto N = 5;

__global__ void gpu_square(float* d_in, float* d_out) {
	int tid = threadIdx.x;
	float temp = d_in[tid];
	d_out[tid] = temp * temp;
}

int main(void)
{
	float h_in[N], h_out[N];
	float* d_in, * d_out;

	hipMalloc((void**)&d_in, N * sizeof(float));
	hipMalloc((void**)&d_out, N * sizeof(float));
	// Ϊ������ֵ
	for (int i = 0; i < N; i++)
	{
		h_in[i] = i;
	}

	hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
	gpu_square << <1, N >> > (d_in, d_out);
	hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

	printf("Square of Number on GPU\n");
	for (int i = 0; i < N; i++)
	{
		printf("The square of %f is %f\n", h_in[i], h_out[i]);
	}
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}
